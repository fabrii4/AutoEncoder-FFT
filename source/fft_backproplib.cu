#include "hip/hip_runtime.h"
#include <ctime>
#include <math.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <chrono>
#include <cstdlib>
#include <fstream>
#include<iomanip>

#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>

using namespace std;


/////////////////////////////////////////////////////////////
//       GPU FUNCTIONS
/////////////////////////////////////////////////////////////


//shift fft spectrum to display zero frequency to the center
__global__ void shift_magnitude(hipfftReal *mag, hipfftReal *mag_s, int dD, int Nx, int Ny)
{
   int Ntot=dD*Nx*Ny;
   int idx = threadIdx.x + blockDim.x*blockIdx.x;
   int d=(int)idx/(Nx*Ny); 
   int i=(int)(idx-d*Nx*Ny)/Ny;
   int j=(int)(idx-d*Nx*Ny-i*Ny);
   int ind;
   if (idx < Ntot)
   {
      if(i<Nx/2 && j<Ny/2) ind=d*Nx*Ny+(Nx/2+i)*Ny+(Ny/2+j);
      if(i<Nx/2 && j>=Ny/2) ind=d*Nx*Ny+(Nx/2+i)*Ny+(j-Ny/2);
      if(i>=Nx/2 && j<Ny/2) ind=d*Nx*Ny+(i-Nx/2)*Ny+(Ny/2+j);
      if(i>=Nx/2 && j>=Ny/2) ind=d*Nx*Ny+(i-Nx/2)*Ny+(j-Ny/2);
      mag_s[idx]=mag[ind];
   }
}

/////////////////////////////////////////////////////////////

//compute fft spectrum with no shift
__global__ void magnitude(hipfftComplex *data, hipfftReal *mag, int dD, int Nx, int Ny)
{
   int Ntot=dD*Nx*Ny;
   int idx = threadIdx.x + blockDim.x*blockIdx.x;
   int d=(int)idx/(Nx*Ny);
   int i=(int)(idx-d*Nx*Ny)/Ny;
   int j=(int)(idx-d*Nx*Ny-i*Ny);
   int Nyr=Ny/2+1;
   int indl=d*Nx*Nyr+i*Nyr+j;
   int indr=d*Nx*Nyr+(Nx-1-i)*Nyr+(2*Nyr-1-j);
   if(idx<Ntot)
   {
      if(j<Nyr) mag[idx]=sqrt(hipCabsf(data[indl])/Ntot);
      else mag[idx]=sqrt(hipCabsf(data[indr])/Ntot);
   }
}

/////////////////////////////////////////////////////////////

//Pooling (crop fft spectrum around zero frequency)
__global__ void pool(hipfftComplex *data, int dD, int Nx, int Ny, int scale)
{
   float l=1./((float)scale*2);
   int Nyr=Ny/2+1;
   int idx = threadIdx.x + blockDim.x*blockIdx.x;
   if (idx < dD*Nx*Nyr)
   {
      int d=(int)idx/(Nx*Nyr);
      int i=(int)(idx-d*Nx*Nyr)/Nyr;
      int j=(int)(idx-d*Nx*Nyr-i*Nyr);
      //if(!((i<l*Nx || i>(1.-l)*Nx) && (j<l*Nyr || j>(1.-l)*Nyr)))
      if(!((i<l*Nx || i>=(1.-l)*Nx) && (j<l*Ny || j>=(1.-l)*Ny)))
         data[idx].x=data[idx].y=0;
   }
}

/////////////////////////////////////////////////////////////

//resize according to spectral pooling (for even Nx Ny Nxs Nys)
__global__ void resize(hipfftComplex *freq_d, hipfftComplex *freqs_d, int dM, int Nx, int Ny, int Nxs, int Nys, float l)
{
   int Nyr=Ny/2+1;
   int Nyrs=Nys/2+1;
   int ind;
   int idx = threadIdx.x + blockDim.x*blockIdx.x;
   if (idx < dM*Nxs*Nyrs)
   {
      int d=(int)idx/(Nxs*Nyrs);
      int i=(int)(idx-d*Nxs*Nyrs)/Nyrs;
      int j=(int)(idx-d*Nxs*Nyrs-i*Nyrs);
      if(Nxs<=Nx)
      {
         if(j<Nyrs-1)
         {
            if(i<Nxs/2) ind=d*Nx*Nyr+i*Nyr+j;
            if(i==Nxs/2) ind=d*Nx*Nyr+Nx/2*Nyr+j;
            if(i>Nxs/2) ind=d*Nx*Nyr+(i+Nx-Nxs)*Nyr+j;
            freqs_d[idx]=freq_d[ind];
         }
         if(j==Nyrs-1)
         {
            if(i<Nxs/2) ind=d*Nx*Nyr+i*Nyr+Nyr-1;
            if(i==Nxs/2) ind=d*Nx*Nyr+Nx/2*Nyr+Nyr-1;
            if(i>Nxs/2) ind=d*Nx*Nyr+(i+Nx-Nxs)*Nyr+Nyr-1;
            freqs_d[idx]=freq_d[ind];
         }
      }
      else
      {
         if(j<Nyr-1)
         {
            if(i<Nx/2)
            {
               ind=d*Nx*Nyr+i*Nyr+j;
               freqs_d[idx]=freq_d[ind];
            }
            else if(i>Nxs-Nx/2) 
            {
               ind=d*Nx*Nyr+(i-Nxs+Nx)*Nyr+j;
               freqs_d[idx]=freq_d[ind];
            }
            else if(i==Nxs/2) 
            {
               ind=d*Nx*Nyr+Nx/2*Nyr+j;
               freqs_d[idx]=freq_d[ind];
            }
         }
         if(j==Nyrs-1)
         {
            if(i<Nx/2)
            {
               ind=d*Nx*Nyr+i*Nyr+Nyr-1;
               freqs_d[idx]=freq_d[ind];
            }
            else if(i>Nxs-Nx/2) 
            {
               ind=d*Nx*Nyr+(i-Nxs+Nx)*Nyr+Nyr-1;
               freqs_d[idx]=freq_d[ind];
            }
            else if(i==Nxs/2) 
            {
               ind=d*Nx*Nyr+Nx/2*Nyr+Nyr-1;
               freqs_d[idx]=freq_d[ind];
            }
         }
      }
      //freqs_d[idx].x/=l;
      //freqs_d[idx].y/=l;
   }
}

/////////////////////////////////////////////////////////////

//convolution (out must be initialized to zero)
__global__ void conv_k(hipfftComplex *in, hipfftComplex *out, hipfftComplex *c, hipfftReal *b, int dM, int dD, int Nx, int Ny)
{
   int Nyr=Ny/2+1;
   int idx = threadIdx.x + blockDim.x*blockIdx.x;
   if (idx < dM*Nx*Nyr)
   {
      int m=(int)idx/(Nx*Nyr);
      int i=(int)(idx-m*Nx*Nyr)/Nyr;
      int j=(int)(idx-m*Nx*Nyr-i*Nyr);
      hipfftComplex out_t;
      out_t.x=0; out_t.y=0;
      for(int d=0; d<dD; d++)
      {
         hipfftComplex in_t=in[d*Nx*Nyr+i*Nyr+j];
         in_t.x/=dM;
         in_t.y/=dM;
         hipfftComplex c_t=c[m*dD*Nx*Nyr+d*Nx*Nyr+i*Nyr+j];
         //out[m*Nx*Nyr+i*Nyr+j].x+=(in_t.x*c_t.x-in_t.y*c_t.y);
         //out[m*Nx*Nyr+i*Nyr+j].y+=(in_t.x*c_t.y+in_t.y*c_t.x);
         out_t.x+=(in_t.x*c_t.x-in_t.y*c_t.y);
         out_t.y+=(in_t.x*c_t.y+in_t.y*c_t.x);
         if(d==0 && i==0 && j==0)
            out_t.x+=b[m]*Nx*Ny;
            //out[m*Nx*Nyr+i*Nyr+j].x+=b[m]*Nx*Ny;
      }
      out[idx]=out_t;
   }
}

__global__ void conv_k1(hipfftComplex *in, hipfftComplex *out, hipfftComplex *c, hipfftReal *b, int dM, int dD, int Nx, int Ny)
{
   extern __shared__ hipfftComplex in_s[];
   extern __shared__ hipfftComplex c_s[];
   int Nyr=Ny/2+1;
   int idx = threadIdx.x + blockDim.x*blockIdx.x;
   if(idx < dM*dD*Nx*Nyr)
   {
      if(idx < dD*Nx*Nyr)
         in_s[idx]=in[idx];
      c_s[idx]=c[idx];
      __syncthreads();
      if (idx < dM*Nx*Nyr)
      {
         int m=(int)idx/(Nx*Nyr);
         int i=(int)(idx-m*Nx*Nyr)/Nyr;
         int j=(int)(idx-m*Nx*Nyr-i*Nyr);
         hipfftComplex out_t;
         out_t.x=0; out_t.y=0;
         for(int d=0; d<dD; d++)
         {
            hipfftComplex in_t=in_s[d*Nx*Nyr+i*Nyr+j];
            in_t.x/=dM;
            in_t.y/=dM;
            hipfftComplex c_t=c_s[m*dD*Nx*Nyr+d*Nx*Nyr+i*Nyr+j];
            //out[m*Nx*Nyr+i*Nyr+j].x+=(in_t.x*c_t.x-in_t.y*c_t.y);
            //out[m*Nx*Nyr+i*Nyr+j].y+=(in_t.x*c_t.y+in_t.y*c_t.x);
            out_t.x+=(in_t.x*c_t.x-in_t.y*c_t.y);
            out_t.y+=(in_t.x*c_t.y+in_t.y*c_t.x);
            if(d==0 && i==0 && j==0)
               out_t.x+=b[m]*Nx*Ny;
               //out[m*Nx*Nyr+i*Nyr+j].x+=b[m]*Nx*Ny;
         }
         out[idx]=out_t;
      }
   }
}

/////////////////////////////////////////////////////////////

//normalize input to convolution to output size
__global__ void normalize( hipfftComplex* freq_d, int dM, int dD, int Nx, int Ny)
{
   int Nyr=Ny/2+1;
   int idx = threadIdx.x + blockDim.x*blockIdx.x;
   if (idx < dD*Nx*Nyr)
   {
      freq_d[idx].x/=dM;
      freq_d[idx].y/=dM;
   }
}

/////////////////////////////////////////////////////////////

//copy hipfftComplex Ntot array to float 2*Ntot array
__global__ void copy_out(hipfftComplex *cfreq_d, float *Cc_d, int dM, int dD, int Nx, int Ny)
{
   int Ntot=dM*dD*Nx*Ny*2;
   int idx = threadIdx.x + blockDim.x*blockIdx.x;
   int m=(int)idx/(dD*Nx*Ny*2); 
   int d=(int)(idx-m*dD*Nx*Ny*2)/(Nx*Ny*2); 
   int i=(int)(idx-m*dD*Nx*Ny*2-d*Nx*Ny*2)/(Ny*2);
   int j=(int)(idx-m*dD*Nx*Ny*2-d*Nx*Ny*2-i*Ny*2)/2;
   if (idx < Ntot)
   {
      int ind=m*dD*Nx*Ny+d*Nx*Ny+i*Ny+j;
      hipfftComplex cfreq_t=cfreq_d[ind];
      float cfreq_val;
      if(idx%2==0) cfreq_val=cfreq_t.x;
      else cfreq_val=cfreq_t.y;
      Cc_d[idx]=cfreq_val;
   }
}


/////////////////////////////////////////////////////////////

//copy float 2*Ntot array to hipfftComplex Ntot array
__global__ void copy_in(float *Cc_d, hipfftComplex *cfreq_d, int dM, int dD, int Nx, int Ny)
{
   int Ntot=dM*dD*Nx*Ny;
   int idx = threadIdx.x + blockDim.x*blockIdx.x;
   int m=(int)idx/(dD*Nx*Ny); 
   int d=(int)(idx-m*dD*Nx*Ny)/(Nx*Ny); 
   int i=(int)(idx-m*dD*Nx*Ny-d*Nx*Ny)/Ny;
   int j=(int)(idx-m*dD*Nx*Ny-d*Nx*Ny-i*Ny);
   if (idx < Ntot)
   {
      cfreq_d[idx].x=Cc_d[m*dD*Nx*Ny*2+d*Nx*Ny*2+i*Ny*2+j*2+0];
      cfreq_d[idx].y=Cc_d[m*dD*Nx*Ny*2+d*Nx*Ny*2+i*Ny*2+j*2+1];
   }
}

/////////////////////////////////////////////////////////////

__device__ void adapt_rate(float& delR, float& delI, float delmax, float dDdXR, float dDdXI, hipfftComplex& ddx, hipfftComplex dx)
{
   delR=delI=delmax;
   float dddxR=dDdXR-ddx.x;
   float dddxI=dDdXI-ddx.y;
   if(dddxR!=0) delR=abs(dx.x/dddxR);
   if(dddxI!=0) delI=abs(dx.y/dddxI);
   if(delR>delmax) delR=delmax;
   if(delI>delmax) delI=delmax;
   ddx.x=dDdXR;
   ddx.y=dDdXI;
}

/////////////////////////////////////////////////////////////

__device__ void adapt_rateR(float& del, float delmax, float dDdX, hipfftReal& ddx, hipfftReal dx)
{
   del=delmax;
   float dddx=dDdX-ddx;
   if(dddx!=0) del=abs(dx/dddx);
   if(del>delmax) del=delmax;
   ddx=dDdX;
}

/////////////////////////////////////////////////////////////

__global__ void backprop_k(hipfftComplex *freq_d, hipfftComplex *ofreq_d, hipfftComplex *cfreq_d, hipfftComplex *ffreq_d, hipfftReal *b_d, hipfftReal *p_d, hipfftComplex *cfreq1_d, hipfftComplex *ffreq1_d, hipfftReal *b1_d, hipfftReal *p1_d, hipfftComplex *dc_d, hipfftComplex *df_d, hipfftReal *db_d, hipfftReal *dp_d, hipfftComplex *ddc_d, hipfftComplex *ddf_d, hipfftReal *ddb_d, hipfftReal *ddp_d, int dM, int dD, int Nx, int Ny)
{
   int Nyr=Ny/2+1;
   float norm=Nx*Ny;
   float n=norm*2*dM*dD*Nx*Ny;
   float delmax=0.5;
   float delR, delI;
   float alpha=0.9;
   int idx = threadIdx.x + blockDim.x*blockIdx.x;
   if(idx<dM*dD*Nx*Nyr)
   {
      int m=(int)idx/(dD*Nx*Nyr);
      int d=(int)(idx-m*dD*Nx*Nyr)/(Nx*Nyr);
      int i=(int)(idx-m*dD*Nx*Nyr-d*Nx*Nyr)/Nyr;
      int j=(int)(idx-m*dD*Nx*Nyr-d*Nx*Nyr-i*Nyr);
      float Norm=n;
      if(j>0 && j<Nyr-1) Norm/=2;
      float sumcRR=0, sumcRI=0, sumcIR=0, sumcII=0;
      float sumfR=0, sumfI=0;
      float sumb=0;
      for(int d1=0;d1<dD;d1++)
      {
         //c derivative sums over d
         int ind=d1*Nx*Nyr+i*Nyr+j;
         int indf=d1*dM*Nx*Nyr+m*Nx*Nyr+i*Nyr+j;
         hipfftComplex ofreq=ofreq_d[ind];
         hipfftComplex freq=freq_d[ind];
         hipfftComplex ffreq=ffreq_d[indf];
         sumcRR+=(ofreq.x-freq.x)*ffreq.x;
         sumcRI+=(ofreq.x-freq.x)*ffreq.y;
         sumcIR+=(ofreq.y-freq.y)*ffreq.x;
         sumcII+=(ofreq.y-freq.y)*ffreq.y;
         //f derivative sums over d
         int indc=m*dD*Nx*Nyr+d1*Nx*Nyr+i*Nyr+j;
         hipfftComplex cfreq=cfreq_d[indc];
         sumfR+=cfreq.x*freq.x-cfreq.y*freq.y;
         sumfI+=cfreq.x*freq.y+cfreq.y*freq.x;
         //b derivative sum over d
         if(i==0 && j==0)
            sumb+=(ofreq.x-freq.x)*ffreq.x+(ofreq.y-freq.y)*ffreq.y;
      }
      //c update
      int ind=d*Nx*Nyr+i*Nyr+j;
      hipfftComplex freq=freq_d[ind];
      float dDR=sumcRR*freq.x-sumcRI*freq.y+sumcIR*freq.y+sumcII*freq.x;
      float dDI=-sumcRR*freq.y-sumcRI*freq.x+sumcIR*freq.x-sumcII*freq.y;
      float dDdCR=dDR/Norm;
      float dDdCI=dDI/Norm;
      hipfftComplex cfreq=cfreq_d[idx];
      hipfftComplex dc=dc_d[idx];
      adapt_rate(delR, delI, delmax, dDdCR, dDdCI, ddc_d[idx], dc);
      dc.x=(1-alpha)*delR*dDdCR/((10<abs(dDdCR))?abs(dDdCR):10)+alpha*dc.x;
      dc.y=(1-alpha)*delI*dDdCI/((10<abs(dDdCI))?abs(dDdCI):10)+alpha*dc.y;
      cfreq1_d[idx].x=cfreq.x-dc.x;
      cfreq1_d[idx].y=cfreq.y-dc.y;
      dc_d[idx]=dc;
      //f update
      int idxf=d*dM*Nx*Nyr+m*Nx*Nyr+i*Nyr+j;
      float b0=0;
      if(i==0 && j==0) 
         b0=b_d[m]*norm;
      hipfftComplex ofreq=ofreq_d[ind];
      float diffR=ofreq.x-freq.x;
      float diffI=ofreq.y-freq.y;
      dDR=diffR*(sumfR+b0)+diffI*sumfI;
      dDI=-diffR*sumfI+diffI*(sumfR+b0);
      float dDdFR=dDR/Norm;
      float dDdFI=dDI/Norm;
      hipfftComplex ffreq=ffreq_d[idxf];
      hipfftComplex df=df_d[idxf];
      adapt_rate(delR, delI, delmax, dDdFR, dDdFI, ddf_d[idxf], df);
      df.x=(1-alpha)*delR*dDdFR/((10<abs(dDdFR))?abs(dDdFR):10)+alpha*df.x;
      df.y=(1-alpha)*delI*dDdFI/((10<abs(dDdFI))?abs(dDdFI):10)+alpha*df.y;
      ffreq1_d[idxf].x=ffreq.x-df.x;
      ffreq1_d[idxf].y=ffreq.y-df.y;
      df_d[idxf]=df;
      //b update
      if(i==0 && j==0 && d==0)
      {
         float dDdB=sumb*norm/Norm;
         hipfftReal db=db_d[m];
         adapt_rateR(delR, delmax, dDdB, ddb_d[m], db);
         db=(1-alpha)*delR*dDdB/((10<abs(dDdB))?abs(dDdB):10)+alpha*db;
         b1_d[m]=b_d[m]-db;
         db_d[m]=db;
      }
      //p update
      if(i==0 && j==0 && m==0)
      {
         float dDdP=(ofreq.x-freq.x)*norm/Norm;
         hipfftReal dp=dp_d[d];
         adapt_rateR(delR, delmax, dDdP, ddp_d[d], dp);
         dp=(1-alpha)*delR*dDdP/((10<abs(dDdP))?abs(dDdP):10)+alpha*dp;
         p1_d[d]=p_d[d]-dp;
         dp_d[d]=dp;
      }
   }
}

/////////////////////////////////////////////////////////////

__global__ void gradient_k(hipfftComplex *freq_d, hipfftComplex *ofreq_d, hipfftComplex *cfreq_d, hipfftComplex *ffreq_d, hipfftReal *b_d, hipfftReal *p_d, hipfftComplex *dc_d, hipfftComplex *df_d, hipfftReal *db_d, hipfftReal *dp_d, int dM, int dD, int Nx, int Ny)
{
   int Nyr=Ny/2+1;
   float norm=Nx*Ny;
   float n=norm*2*dM*dD*Nx*Ny;
   int idx = threadIdx.x + blockDim.x*blockIdx.x;
   if(idx<dM*dD*Nx*Nyr)
   {
      int m=(int)idx/(dD*Nx*Nyr);
      int d=(int)(idx-m*dD*Nx*Nyr)/(Nx*Nyr);
      int i=(int)(idx-m*dD*Nx*Nyr-d*Nx*Nyr)/Nyr;
      int j=(int)(idx-m*dD*Nx*Nyr-d*Nx*Nyr-i*Nyr);
      float Norm=n;
      //if(j>0 && (Ny%2==0)?(j<Nyr-1):(j<Nyr)) Norm/=2; 
      float sumcRR=0, sumcRI=0, sumcIR=0, sumcII=0;
      float sumfR=0, sumfI=0;
      float sumb=0;
      for(int d1=0;d1<dD;d1++)
      {
         //c derivative sums over d
         int ind=d1*Nx*Nyr+i*Nyr+j;
         int indf=d1*dM*Nx*Nyr+m*Nx*Nyr+i*Nyr+j;
         hipfftComplex ofreq=ofreq_d[ind];
         hipfftComplex freq=freq_d[ind];
         hipfftComplex ffreq=ffreq_d[indf];
         sumcRR+=(ofreq.x-freq.x)*ffreq.x;
         sumcRI+=(ofreq.x-freq.x)*ffreq.y;
         sumcIR+=(ofreq.y-freq.y)*ffreq.x;
         sumcII+=(ofreq.y-freq.y)*ffreq.y;
         //f derivative sums over d
         int indc=m*dD*Nx*Nyr+d1*Nx*Nyr+i*Nyr+j;
         hipfftComplex cfreq=cfreq_d[indc];
         sumfR+=cfreq.x*freq.x-cfreq.y*freq.y;
         sumfI+=cfreq.x*freq.y+cfreq.y*freq.x;
         //b derivative sum over d
         if(i==0 && j==0)
            sumb+=(ofreq.x-freq.x)*ffreq.x+(ofreq.y-freq.y)*ffreq.y;
      }
      //c update
      int ind=d*Nx*Nyr+i*Nyr+j;
      hipfftComplex freq=freq_d[ind];
      float dDR=sumcRR*freq.x-sumcRI*freq.y+sumcIR*freq.y+sumcII*freq.x;
      float dDI=-sumcRR*freq.y-sumcRI*freq.x+sumcIR*freq.x-sumcII*freq.y;
      float dDdCR=dDR/Norm;
      float dDdCI=dDI/Norm;
      hipfftComplex dc;
      dc.x=dDdCR;
      dc.y=dDdCI;
      dc_d[idx]=dc;
      //f update
      int idxf=d*dM*Nx*Nyr+m*Nx*Nyr+i*Nyr+j;
      float b0=0;
      if(i==0 && j==0) 
         b0=b_d[m]*norm;
      hipfftComplex ofreq=ofreq_d[ind];
      float diffR=ofreq.x-freq.x;
      float diffI=ofreq.y-freq.y;
      dDR=diffR*(sumfR+b0)+diffI*sumfI;
      dDI=-diffR*sumfI+diffI*(sumfR+b0);
      float dDdFR=dDR/Norm;
      float dDdFI=dDI/Norm;
      hipfftComplex df;
      df.x=dDdFR;
      df.y=dDdFI;
      df_d[idxf]=df;
      //b update
      if(i==0 && j==0 && d==0)
      {
         float dDdB=sumb*norm/Norm;
         db_d[m]=dDdB;
      }
      //p update
      if(i==0 && j==0 && m==0)
      {
         float dDdP=(ofreq.x-freq.x)*norm/Norm;
         dp_d[d]=dDdP;
      }
   }
}


/////////////////////////////////////////////////////////////

//calculate mse
__global__ void calc_mse(hipfftComplex *freq_d, hipfftComplex *ofreq_d, int dD, int Nx, int Ny, float *Cout)
{
   int Nyr=Ny/2+1;
   float norm=dD*Nx*Ny;
   int idx = threadIdx.x + blockDim.x*blockIdx.x;
   if(idx<dD*Nx*Nyr)
   {
      int d=(int)(idx)/(Nx*Nyr);
      int i=(int)(idx-d*Nx*Nyr)/Nyr;
      int j=(int)(idx-d*Nx*Nyr-i*Nyr);
      float fR=freq_d[idx].x;
      float fI=freq_d[idx].y;
      float ofR=ofreq_d[idx].x;
      float ofI=ofreq_d[idx].y;
      float n=norm;
      if(j>0 && j<Nyr-1) n/=2;
      Cout[idx]=((fR-ofR)*(fR-ofR)+(fI-ofI)*(fI-ofI))/n;
   }
}

/////////////////////////////////////////////////////////////

//__global__ void shrink_k<<<blocks,threads>>>(hipfftReal *dc_d, hipfftReal *dck_d, int dM, int dD, int Nx, int Ny, int Nk, int Nl)
//{
//   int idx = threadIdx.x + blockDim.x*blockIdx.x;
//   if(idx<dM*dD*Nx*Ny)
//   {
//      int m=(int)idx/(dD*Nx*Ny);
//      int d=(int)(idx-m*dD*Nx*Ny)/(Nx*Ny);
//      int i=(int)(idx-m*dD*Nx*Ny-d*Nx*Ny)/Ny;
//      int j=(int)(idx-m*dD*Nx*Ny-d*Nx*Ny-i*Ny);
//      if(i<Nk/2+1 && j<Nl/2+1)
//      {
//         int idk=m*dD*Nk*Nl+d*Nk*Nl+(i+Nk/2)*Nk+(j+Nl/2);
//         dck_d[idk]=dc_d[idx];
//      }
//      if(i>=Nx-Nk/2 && j<Nl/2+1)
//      {
//         int idk=m*dD*Nk*Nl+d*Nk*Nl+(i-Nx+Nk/2)*Nk+(j+Nl/2);
//         dck_d[idk]=dc_d[idx];
//      }
//      if(i<Nk/2+1 && j>=Ny-Nl/2)
//      {
//         int idk=m*dD*Nk*Nl+d*Nk*Nl+(i+Nk/2)*Nk+(j-Ny+Nl/2);
//         dck_d[idk]=dc_d[idx];
//      }
//      if(i>=Nx-Nk/2 && j>=Ny-Nl/2)
//      {
//         int idk=m*dD*Nk*Nl+d*Nk*Nl+(i-Nx+Nk/2)*Nk+(j-Ny+Nl/2);
//         dck_d[idk]=dc_d[idx];
//      }
//   }
//}

//shrink padded kernel to Nk*Nl size
__global__ void shrink_k(hipfftReal *dc_d, hipfftReal *dck_d, int dM, int dD, int Nx, int Ny, int Nk, int Nl)
{
   int idk = threadIdx.x + blockDim.x*blockIdx.x;
   if(idk<dM*dD*Nk*Nl)
   {
      int m=(int)idk/(dD*Nk*Nl);
      int d=(int)(idk-m*dD*Nk*Nl)/(Nk*Nl);
      int k=(int)(idk-m*dD*Nk*Nl-d*Nk*Nl)/Nl;
      int l=(int)(idk-m*dD*Nk*Nl-d*Nk*Nl-k*Nl);
      if(k>=Nk/2 && l>=Nl/2)
      {
         int idx=m*dD*Nx*Ny+d*Nx*Ny+(k-Nk/2)*Ny+(l-Nl/2);
         dck_d[idk]=dc_d[idx];
      }
      if(k<Nk/2 && l>=Nl/2)
      {
         int idx=m*dD*Nx*Ny+d*Nx*Ny+(k+Nx-Nk/2)*Ny+(l-Nl/2);
         dck_d[idk]=dc_d[idx];
      }
      if(k>=Nk/2 && l<Nl/2)
      {
         int idx=m*dD*Nx*Ny+d*Nx*Ny+(k-Nk/2)*Ny+(l+Ny-Nl/2);
         dck_d[idk]=dc_d[idx];
      }
      if(k<Nk/2 && l<Nl/2)
      {
         int idx=m*dD*Nx*Ny+d*Nx*Ny+(k+Nx-Nk/2)*Ny+(l+Ny-Nl/2);
         dck_d[idk]=dc_d[idx];
      }
   }
}

/////////////////////////////////////////////////////////////

//expand kernel to input size Nx*Ny
__global__ void pad_k(hipfftReal *dck_d, hipfftReal *dc_d, int dM, int dD, int Nx, int Ny, int Nk, int Nl)
{
   int idk = threadIdx.x + blockDim.x*blockIdx.x;
   if(idk<dM*dD*Nk*Nl)
   {
      int m=(int)idk/(dD*Nk*Nl);
      int d=(int)(idk-m*dD*Nk*Nl)/(Nk*Nl);
      int k=(int)(idk-m*dD*Nk*Nl-d*Nk*Nl)/Nl;
      int l=(int)(idk-m*dD*Nk*Nl-d*Nk*Nl-k*Nl);
      if(k>=Nk/2 && l>=Nl/2)
      {
         int idx=m*dD*Nx*Ny+d*Nx*Ny+(k-Nk/2)*Ny+(l-Nl/2);
         dc_d[idx]=dck_d[idk];
      }
      if(k<Nk/2 && l>=Nl/2)
      {
         int idx=m*dD*Nx*Ny+d*Nx*Ny+(k+Nx-Nk/2)*Ny+(l-Nl/2);
         dc_d[idx]=dck_d[idk];
      }
      if(k>=Nk/2 && l<Nl/2)
      {
         int idx=m*dD*Nx*Ny+d*Nx*Ny+(k-Nk/2)*Ny+(l+Ny-Nl/2);
         dc_d[idx]=dck_d[idk];
      }
      if(k<Nk/2 && l<Nl/2)
      {
         int idx=m*dD*Nx*Ny+d*Nx*Ny+(k+Nx-Nk/2)*Ny+(l+Ny-Nl/2);
         dc_d[idx]=dck_d[idk];
      }
   }
}

/////////////////////////////////////////////////////////////

//compute backpropagation in coordinate space for kernel weights c, f and biases b, p
__global__ void backprop_d(hipfftReal *c_d, hipfftReal *f_d, hipfftReal *b_d, hipfftReal *p_d, hipfftReal *dck_d, hipfftReal *dfk_d, hipfftReal *db_d, hipfftReal *dp_d, hipfftReal *Dc_d, hipfftReal *Df_d, hipfftReal *Db_d, hipfftReal *Dp_d, hipfftReal *ddc, hipfftReal *ddf, hipfftReal *ddb, hipfftReal *ddp, int dD, int dM, int Nk, int Nl, float del)
{
   //float del=0.00001, delmax=0.1;
   float alpha=0.9;
   int idk = threadIdx.x + blockDim.x*blockIdx.x;
   if(idk<dM*dD*Nk*Nl)
   {
      //c update
      float dDdC=dck_d[idk];
      float dc=Dc_d[idk];
//      adapt_rateR(del, delmax, dDdC, ddc[idk], dc);
      float Dc=(1-alpha)*del*dDdC/((10<abs(dDdC))?abs(dDdC):10)+alpha*dc;
      c_d[idk]+= -Dc;
      Dc_d[idk]=Dc;
//      c_d[idk]+= -del*dck_d[idk];
      //f update
      float dDdF=dfk_d[idk];
      float df=Df_d[idk];
//      adapt_rateR(del, delmax, dDdF, ddf[idk], df);
      float Df=(1-alpha)*del*dDdF/((10<abs(dDdF))?abs(dDdF):10)+alpha*df;
      f_d[idk]+= -Df;
      Df_d[idk]=Df;
//      f_d[idk]+= -del*dfk_d[idk];
      //b update
      if(idk<dM)
      {
         float dDdB=db_d[idk];
         float db=Db_d[idk];
//         adapt_rateR(del, delmax, dDdB, ddb[idk], db);
         float Db=(1-alpha)*del*dDdB/((10<abs(dDdB))?abs(dDdB):10)+alpha*db;
         b_d[idk]+= -Db;
         Db_d[idk]=Db;
//         b_d[idk]+= -del*db_d[idk];
      }
      //p update
      if(idk<dD)
      {
         float dDdP=dp_d[idk];
         float dp=Dp_d[idk];
//         adapt_rateR(del, delmax, dDdP, ddp[idk], dp);
         float Dp=(1-alpha)*del*dDdP/((10<abs(dDdP))?abs(dDdP):10)+alpha*dp;
         p_d[idk]+= -Dp;
         Dp_d[idk]=Dp;
//         p_d[idk]+= -del*dp_d[idk];
      }
   }

}

/////////////////////////////////////////////////////////////

//compute backpropagation in coordinate space for kernel weights c, f and biases b, p in multiobjective optimization (D=w0*D0-w1*D1)
__global__ void backprop_double(hipfftReal *c_d, hipfftReal *f_d, hipfftReal *b_d, hipfftReal *p_d, hipfftReal *dck_d, hipfftReal *dfk_d, hipfftReal *db_d, hipfftReal *dp_d, hipfftReal *Dc_d, hipfftReal *Df_d, hipfftReal *Db_d, hipfftReal *Dp_d, hipfftReal *ddc, hipfftReal *ddf, hipfftReal *ddb, hipfftReal *ddp, hipfftReal *cd_d, hipfftReal *fd_d, hipfftReal *bd_d, hipfftReal *pd_d, int dD, int dM, int Nk, int Nl, float del, float w0, float w1)
{
   //float del=0.00001, delmax=0.1;
   float alpha=0.9;
   int idk = threadIdx.x + blockDim.x*blockIdx.x;
   if(idk<dM*dD*Nk*Nl)
   {
      //c update
      float dDdC=w0*dck_d[idk]-w1*cd_d[idk];
      float dc=Dc_d[idk];
//      adapt_rateR(del, delmax, dDdC, ddc[idk], dc);
      float Dc=(1-alpha)*del*dDdC/((10<abs(dDdC))?abs(dDdC):10)+alpha*dc;
      c_d[idk]+= -Dc;
      Dc_d[idk]=Dc;
//      c_d[idk]+= -del*dck_d[idk];
      //f update
      float dDdF=w0*dfk_d[idk]-w1*fd_d[idk];
      float df=Df_d[idk];
//      adapt_rateR(del, delmax, dDdF, ddf[idk], df);
      float Df=(1-alpha)*del*dDdF/((10<abs(dDdF))?abs(dDdF):10)+alpha*df;
      f_d[idk]+= -Df;
      Df_d[idk]=Df;
//      f_d[idk]+= -del*dfk_d[idk];
      //b update
      if(idk<dM)
      {
         float dDdB=w0*db_d[idk]-w1*bd_d[idk];
         float db=Db_d[idk];
//         adapt_rateR(del, delmax, dDdB, ddb[idk], db);
         float Db=(1-alpha)*del*dDdB/((10<abs(dDdB))?abs(dDdB):10)+alpha*db;
         b_d[idk]+= -Db;
         Db_d[idk]=Db;
//         b_d[idk]+= -del*db_d[idk];
      }
      //p update
      if(idk<dD)
      {
         float dDdP=w0*dp_d[idk]-w1*pd_d[idk];
         float dp=Dp_d[idk];
//         adapt_rateR(del, delmax, dDdP, ddp[idk], dp);
         float Dp=(1-alpha)*del*dDdP/((10<abs(dDdP))?abs(dDdP):10)+alpha*dp;
         p_d[idk]+= -Dp;
         Dp_d[idk]=Dp;
//         p_d[idk]+= -del*dp_d[idk];
      }
   }

}

/////////////////////////////////////////////////////////////

//compute gradient for kernel distance to use in multiobjective optimization
__global__ void gradient_diff(hipfftReal *cd_d, hipfftReal *fd_d, hipfftReal *bd_d, hipfftReal *pd_d, hipfftReal *c_d, hipfftReal *f_d, hipfftReal *b_d, hipfftReal * p_d, int dD, int dM, int Nk, int Nl)
{

   int idk = threadIdx.x + blockDim.x*blockIdx.x;
   if(idk < dD*dM*Nk*Nl)
   {
      int m=(int)idk/(dD*Nk*Nl);
      int d=(int)(idk-m*dD*Nk*Nl)/(Nk*Nl);
      int k=(int)(idk-m*dD*Nk*Nl-d*Nk*Nl)/Nl;
      int l=(int)(idk-m*dD*Nk*Nl-d*Nk*Nl-k*Nl);
      float sum_c=0, sum_f=0, sum_b=0, sum_p=0;
      for(int m1=0; m1<dM; m1++)
      {
         for(int d1=0; d1<dD; d1++)
         {
            if(m1!=m && d1!=d)
            {
               float den_c=0, den_f=0;
               for(int k1=0; k1<Nk; k1++)
               {
                  for(int l1=0; l1<Nl; l1++)
                  {
                     float den_c_t=c_d[m*dD*Nk*Nl+d*Nk*Nl+k1*Nl+l1]-c_d[m1*dD*Nk*Nl+d1*Nk*Nl+k1*Nl+l1];
                     float den_f_t=f_d[d*dM*Nk*Nl+m*Nk*Nl+k1*Nl+l1]-f_d[d1*dM*Nk*Nl+m1*Nk*Nl+k1*Nl+l1];
                     den_c+=den_c_t*den_c_t;
                     den_f+=den_f_t*den_f_t;
                  }
               }
               //den_c*=den_c;
               //den_f*=den_f;
               sum_c+=(c_d[m*dD*Nk*Nl+d*Nk*Nl+k*Nl+l]-c_d[m1*dD*Nk*Nl+d1*Nk*Nl+k*Nl+l])/den_c;
               sum_f+=(f_d[d*dM*Nk*Nl+m*Nk*Nl+k*Nl+l]-f_d[d1*dM*Nk*Nl+m1*Nk*Nl+k*Nl+l])/den_f;
            }
            if(m1==0 && d1!=d)
               sum_p+=1./(p_d[d]-p_d[d1]);
         }
         if(m1!=m)
            sum_b+=1./(b_d[m]-b_d[m1]);
      }
      cd_d[m*dD*Nk*Nl+d*Nk*Nl+k*Nl+l]=sum_c;
      fd_d[d*dM*Nk*Nl+m*Nk*Nl+k*Nl+l]=sum_f;
      bd_d[m]=sum_b;
      pd_d[d]=sum_p;
   }
}





/////////////////////////////////////////////////////////////
//       HOST CUDA FUNCTIONS
/////////////////////////////////////////////////////////////

//compute direct fft
void fft(vector<vector<vector<float> > >& in, hipfftComplex *freq_d)
{
   int dD=in.size();
   int Nx=in[0].size();
   int Ny=in[0][0].size();
   int Ntot=dD*Nx*Ny;

   // cuFFT 2D plans for image FFT
   hipfftHandle f_plan;
   int rank = 2;
   int n[2] = {Nx, Ny};
   int idist = Nx*Ny, odist = Nx*(Ny/2+1);
   int inembed[] = {Nx, Ny};
   int onembed[] = {Nx, Ny/2+1};
   int istride = 1, ostride = 1;
   hipfftPlanMany(&f_plan,rank,n,inembed,istride,idist,onembed,ostride,odist,HIPFFT_R2C,dD);

   //input vectors
   hipfftReal *in_h, *in_d;
   hipHostMalloc((void **) &in_h, Ntot*sizeof(hipfftReal));
   hipMalloc(&in_d, Ntot*sizeof(hipfftReal));
   for(int i=0;i<Nx;i++)
   {
      for(int j=0;j<Ny;j++)
      {
         for(int d=0;d<dD;d++)
            in_h[d*Nx*Ny+i*Ny+j]=(hipfftReal)in[d][i][j];
      }
   }
   hipMemcpy(in_d, in_h, Ntot*sizeof(hipfftReal), hipMemcpyHostToDevice);

   //Compute Forward FFT
   hipfftExecR2C(f_plan, in_d, freq_d);

   hipfftDestroy(f_plan);
   hipFree(in_d);
   hipHostFree(in_h);
}

/////////////////////////////////////////////////////////////

//compute inverse fft
void fft_inv(hipfftComplex *freq_d, vector<vector<vector<float> > >& out)
{
   int dM=out.size();
   int Nx=out[0].size();
   int Ny=out[0][0].size();
   int Ntot=dM*Nx*Ny;

   // cuFFT 2D plans for image FFT
   hipfftHandle i_plan;
   int rank = 2;
   int n[2] = {Nx, Ny};
   int idist = Nx*Ny, odist = Nx*(Ny/2+1);
   int inembed[] = {Nx, Ny};
   int onembed[] = {Nx, Ny/2+1};
   int istride = 1, ostride = 1;
   hipfftPlanMany(&i_plan,rank,n,onembed,ostride,odist,inembed,istride,idist,HIPFFT_C2R,dM);

   //output vector
   hipfftReal *out_h, *out_d;
   hipHostMalloc(&out_h, Ntot*sizeof(hipfftReal));
   hipMalloc(&out_d, Ntot*sizeof(hipfftReal));

   //Compute Inverse FFT
   hipfftExecC2R(i_plan, freq_d, out_d);

   float norm=1./(Nx*Ny);
   hipMemcpy(out_h, out_d, Ntot*sizeof(hipfftReal), hipMemcpyDeviceToHost);

   for(int i=0;i<Nx;i++)
   {
      for(int j=0;j<Ny;j++)
      {
         for(int m=0;m<dM;m++)
         {
            int ind=m*Nx*Ny+i*Ny+j;
            out[m][i][j]=(float)out_h[ind]*norm;
            //out[m][i][j]=max(0.,min(out[m][i][j],255.));
            //cout<<out[d][i][j]<<" ";
         }
      }
   }

//      cout<<"N16inv={";
//      for(int m=0;m<dM;m++)
//      {
//         for(int i=0;i<Nx;i++)
//         {
//            for(int j=0;j<Ny;j++)
//            {
//               cout<<out[m][i][j]<<",";
//            }
//         }
//      }
//      cout<<"};"<<endl<<endl;

   hipfftDestroy(i_plan);
   hipHostFree(out_h);
   hipFree(out_d);
}

/////////////////////////////////////////////////////////////

//compute fft for convolutional layer
void kfft(vector<vector<vector<vector<float> > > >& c, vector<float>& b, hipfftComplex *cfreq_d, hipfftReal *b_d)
{
   int dM=c.size();
   int dD=c[0].size();
   int Nk=c[0][0].size();
   int Nl=c[0][0][0].size();
   int Ntot=dM*dD*Nk*Nl;

   // cuFFT 2D plans for kernel FFT
   hipfftHandle f_plan;
   int rank = 2;
   int n[2] = {Nk, Nl};
   int idist = Nk*Nl, odist = Nk*(Nl/2+1);
   int inembed[] = {Nk, Nl};
   int onembed[] = {Nk, Nl/2+1};
   int istride = 1, ostride = 1;
   hipfftPlanMany(&f_plan,rank,n,inembed,istride,idist,onembed,ostride,odist,HIPFFT_R2C,dM*dD);

   //conv kernel
   hipfftReal *c_h, *c_d, *b_h;
   hipHostMalloc((void **) &c_h, Ntot*sizeof(hipfftReal));
   hipHostMalloc((void **) &b_h, dM*sizeof(hipfftReal));
   hipMalloc(&c_d, Ntot*sizeof(hipfftReal));
   for(int m=0;m<dM;m++)
   {
      for(int d=0;d<dD;d++)
      {
         for(int i=0;i<Nk;i++)
         {
            for(int j=0;j<Nl;j++)
            {
              c_h[m*dD*Nk*Nl+d*Nk*Nl+i*Nl+j]=(hipfftReal)c[m][d][i][j];
            }
         }
      }
      b_h[m]=(hipfftReal)b[m];
   }
   hipMemcpy(c_d, c_h, Ntot*sizeof(hipfftReal), hipMemcpyHostToDevice);
   hipMemcpy(b_d, b_h, dM*sizeof(hipfftReal), hipMemcpyHostToDevice);

   //Compute Forward FFT kernel
   hipfftExecR2C(f_plan, c_d, cfreq_d);

   hipfftDestroy(f_plan);
   hipFree(c_d);
   hipHostFree(c_h);
   hipHostFree(b_h);
}

/////////////////////////////////////////////////////////////

//compute inverse fft for convolutional layer
void kfft_inv(hipfftComplex *cfreq_d, hipfftReal *b_d, vector<vector<vector<vector<float> > > >& c, vector<float>& b)
{
   int dM=c.size();
   int dD=c[0].size();
   int Nk=c[0][0].size();
   int Nl=c[0][0][0].size();
   int Ntot=dM*dD*Nk*Nl;

   // cuFFT 2D plans for kernel FFT
   hipfftHandle i_plan;
   int rank = 2;
   int n[2] = {Nk, Nl};
   int idist = Nk*Nl, odist = Nk*(Nl/2+1);
   int inembed[] = {Nk, Nl};
   int onembed[] = {Nk, Nl/2+1};
   int istride = 1, ostride = 1;
   hipfftPlanMany(&i_plan,rank,n,onembed,ostride,odist,inembed,istride,idist,HIPFFT_C2R,dM*dD);

   //output vector
   hipfftReal *c_h, *c_d, *b_h;
   hipHostMalloc(&c_h, Ntot*sizeof(hipfftReal));
   hipHostMalloc(&b_h, dM*sizeof(hipfftReal));
   hipMalloc(&c_d, Ntot*sizeof(hipfftReal));

   //Compute Inverse FFT
   hipfftExecC2R(i_plan, cfreq_d, c_d);

   float norm=1./(Nk*Nl);
   hipMemcpy(c_h, c_d, Ntot*sizeof(hipfftReal), hipMemcpyDeviceToHost);
   hipMemcpy(b_h, b_d, dM*sizeof(hipfftReal), hipMemcpyDeviceToHost);
   for(int m=0;m<dM;m++)
   {
      for(int d=0;d<dD;d++)
      {
         for(int i=0;i<Nk;i++)
         {
            for(int j=0;j<Nl;j++)
            {
               int ind=m*dD*Nk*Nl+d*Nk*Nl+i*Nl+j;
               c[m][d][i][j]=(float)c_h[ind]*norm;
            }
         }
      }
      b[m]=(float)b_h[m];
   }
   hipfftDestroy(i_plan);
   hipFree(c_d);
   hipHostFree(c_h);
   hipHostFree(b_h);
}

/////////////////////////////////////////////////////////////

//Pooling in fft space
void pool_fft(hipfftComplex * &freq_d, int dD, int& Nx, int& Ny, int scale)
{
   if(scale!=1)
   {
      //crop spectrum around zero frequency
      float l=(float)scale;
      if(scale<0) l=-1./((float)scale);
      //resize spectrum
      int Nxs=Nx/l;
      int Nys=Ny/l;
      int threads=256;
      int blocks=(dD*Nxs*(Nys/2+1))/threads+1;
      hipfftComplex *freqs_d;
      hipMalloc(&freqs_d, dD*Nxs*(Nys/2+1)*sizeof(hipfftComplex));
      //if(Nxs>Nx)
         hipMemset(freqs_d, 0, dD*Nxs*(Nys/2+1)*sizeof(hipfftComplex));

      resize<<<blocks,threads>>>(freq_d, freqs_d, dD, Nx, Ny, Nxs, Nys, l);

      hipFree(freq_d);
      hipMalloc(&freq_d, dD*Nxs*(Nys/2+1)*sizeof(hipfftComplex));
      hipMemcpy(freq_d, freqs_d, dD*Nxs*(Nys/2+1)*sizeof(hipfftComplex), 
                  hipMemcpyDeviceToDevice);
      hipFree(freqs_d);
      Nx=Nxs;
      Ny=Nys;
   }
}

/////////////////////////////////////////////////////////////

//compute convolution in fft space
void conv_fft(hipfftComplex *freq_d, hipfftComplex *ofreq_d, hipfftComplex *cfreq_d, hipfftReal *b_d, int dM, int dD, int Nx, int Ny)
{
   hipMemset(ofreq_d, 0, dM*Nx*(Ny/2+1)*sizeof(hipfftComplex));
   int threads=256;
   int blocks=dM*Nx*(Ny/2+1)/threads+1;
   conv_k<<<blocks,threads>>>(freq_d, ofreq_d, cfreq_d, b_d, dM, dD, Nx, Ny);
}

/////////////////////////////////////////////////////////////

//Pad kernel to input size 
void kernel_pad(vector<vector<vector<vector<float> > > >& c, vector<vector<vector<vector<float> > > >& c_pad, int Nx, int Ny)
{
   int dM=c.size();
   int dD=c[0].size();
   int Nk=c[0][0].size();
   int Nl=c[0][0][0].size();
   vector<vector<vector<vector<float> > > > c_new(dM, vector<vector<vector<float> > >(dD, vector<vector<float> >(Nx, vector<float>(Ny))));
   // Copy original kernel to padded kernel
   for (int m = 0; m < dM; m++) 
   {
      for (int d = 0; d < dD; d++) 
      {
         for (int k = 0; k < Nx; k++) 
         {
            for (int l = 0; l < Ny; l++)
            {
               if(k>=0 && k<=Nk/2 && l>=0 && l<=Nl/2)
               {
                  int ik=Nk/2+k;
                  int il=Nl/2+l;
                  c_new[m][d][k][l] = c[m][d][ik][il];
               }
               else if(k>=Nx-Nk/2 && k<Nx && l>=0 && l<=Nl/2)
               {
                  int ik=k-(Nx-Nk/2);
                  int il=Nl/2+l;
                  c_new[m][d][k][l] = c[m][d][ik][il];
               }
               else if(k>=0 && k<=Nk/2 && l>=Ny-Nl/2 && l<Ny)
               {
                  int ik=Nk/2+k;
                  int il=l-(Ny-Nl/2);
                  c_new[m][d][k][l] = c[m][d][ik][il];
               }
               else if(k>=Nx-Nk/2 && k<Nx && l>=Ny-Nl/2 && l<Ny)
               {
                  int ik=k-(Nx-Nk/2);
                  int il=l-(Ny-Nl/2);
                  c_new[m][d][k][l] = c[m][d][ik][il];
               }
               else c_new[m][d][k][l] = 0;
            }
         }
      }
   }
   c_pad=c_new;
}

/////////////////////////////////////////////////////////////

//Shrink padded kernel to original size 
void kernel_invpad(vector<vector<vector<vector<float> > > >& c_pad, vector<vector<vector<vector<float> > > >& c, int Nx, int Ny)
{
   int dM=c.size();
   int dD=c[0].size();
   int Nk=c[0][0].size();
   int Nl=c[0][0][0].size();
   // Copy original kernel to padded kernel
   for (int m = 0; m < dM; m++) 
   {
      for (int d = 0; d < dD; d++) 
      {
         for (int k = 0; k < Nx; k++) 
         {
            for (int l = 0; l < Ny; l++)
            {
               if(k>=0 && k<=Nk/2 && l>=0 && l<=Nl/2)
               {
                  int ik=Nk/2+k;
                  int il=Nl/2+l;
                  c[m][d][ik][il] = c_pad[m][d][k][l];
               }
               else if(k>=Nx-Nk/2 && k<Nx && l>=0 && l<=Nl/2)
               {
                  int ik=k-(Nx-Nk/2);
                  int il=Nl/2+l;
                  c[m][d][ik][il] = c_pad[m][d][k][l];
               }
               else if(k>=0 && k<=Nk/2 && l>=Ny-Nl/2 && l<Ny)
               {
                  int ik=Nk/2+k;
                  int il=l-(Ny-Nl/2);
                  c[m][d][ik][il] = c_pad[m][d][k][l];
               }
               else if(k>=Nx-Nk/2 && k<Nx && l>=Ny-Nl/2 && l<Ny)
               {
                  int ik=k-(Nx-Nk/2);
                  int il=l-(Ny-Nl/2);
                  c[m][d][ik][il] = c_pad[m][d][k][l];
               }
            }
         }
      }
   }
}

/////////////////////////////////////////////////////////////

//store convolutional kernels (in fourier space) from device to vector
void store_cfreq(hipfftComplex *cfreq_d, vector<float>& c_freq, int dM, int dD, int Nx, int Nyr)
{
   thrust::host_vector<float> c_h(dM*dD*Nx*Nyr*2);
   thrust::device_vector<float> c_d(c_h);
   float* Cc_d=thrust::raw_pointer_cast(&c_d[0]);
   int threads=256;
   int blocks=2*dM*dD*Nx*Nyr/threads+1;
   copy_out<<<blocks,threads>>>(cfreq_d, Cc_d, dM, dD, Nx, Nyr);
   thrust::copy(c_d.begin(), c_d.end(), c_freq.begin());
}

/////////////////////////////////////////////////////////////

//load convolutional kernels (in fourier space) to device
void load_cfreq(vector<float>& c_freq, vector<float>& net_b, hipfftComplex *cfreq_d, hipfftReal *b_d, int dM, int dD, int Nx, int Nyr)
{
   thrust::device_vector<float> c_d(c_freq);
   thrust::device_vector<hipfftReal> netb_d(net_b);
   float* Cc_d=thrust::raw_pointer_cast(&c_d[0]);
   hipfftReal* Cb_d=thrust::raw_pointer_cast(&netb_d[0]);
   int threads=256;
   int blocks=dM*dD*Nx*Nyr/threads+1;
   copy_in<<<blocks,threads>>>(Cc_d, cfreq_d, dM, dD, Nx, Nyr);
   hipMemcpy(b_d, Cb_d, dM*sizeof(hipfftReal), hipMemcpyDeviceToDevice);
}

/////////////////////////////////////////////////////////////

//store or load fft transformed convolutional kernels
void StoreLoad_cfreq(vector<vector<float> >& net_cfreq, vector<vector<vector<vector<vector<float> > > > >& net_c, vector<vector<float> >& net_b, hipfftComplex *cfreq_d, hipfftReal *b_d, int dM, int dD, int Nx, int Ny, int n)
{
         if(net_cfreq.size() < net_c.size()) //cfreq not yet computed and stored
         {
            vector<vector<vector<vector<float> > > > c_pad;
            kernel_pad(net_c[n], c_pad, Nx, Ny);
            kfft(c_pad, net_b[n], cfreq_d, b_d);
            //store fft kernel to host
            vector<float> c_freq(dM*dD*Nx*(Ny/2+1)*2);
            store_cfreq(cfreq_d, c_freq, dM, dD, Nx, Ny/2+1);
            if(n==net_cfreq.size())
               net_cfreq.push_back(c_freq);
         }
         else //copy stored fft kernel to device
            load_cfreq(net_cfreq[n], net_b[n], cfreq_d, b_d, dM, dD, Nx, Ny/2+1);
}

/////////////////////////////////////////////////////////////

//export fft convolutional kernels to standard convolutional kernels
void export_cfreq(vector<vector<vector<vector<float> > > >& c, vector<float>& b, hipfftComplex *cfreq_d, hipfftReal *b_d, int dM, int dD, int Nx, int Ny)
{
   vector<vector<vector<vector<float> > > > c_pad(dM, vector<vector<vector<float> > >(dD, 
                                            vector<vector<float> >(Nx, vector<float>(Ny))));
   kfft_inv(cfreq_d, b_d, c_pad, b);
   kernel_invpad(c_pad, c, Nx, Ny);
}


/////////////////////////////////////////////////////////////

//calculate mse fft
float mse_fft(hipfftComplex *freq_d, hipfftComplex *ofreq_d, int dM, int dD, int Nx, int Ny)
{
   thrust::device_vector<hipfftReal> mse_d(dD*Nx*(Ny/2+1));
   hipfftReal* Cmse=thrust::raw_pointer_cast(&mse_d[0]);
   if(true)
   {
      int threads=256;
      int blocks=(dD*Nx*(Ny/2+1))/threads+1;
      calc_mse<<<blocks,threads>>>(freq_d, ofreq_d, dD, Nx, Ny, Cmse);
   }
   float vmse = thrust::reduce(mse_d.begin(), mse_d.end());
   float norm=2*dM*Nx*Ny;
   vmse/=norm;
   return vmse;
}

/////////////////////////////////////////////////////////////

//backpropagation in coordinate space
void backprop(hipfftReal *c_d, hipfftReal *f_d, hipfftComplex *cfreq_d, hipfftComplex *ffreq_d, hipfftReal *b_d, hipfftReal *p_d, hipfftComplex *fdc_d, hipfftComplex *fdf_d, hipfftReal *db_d, hipfftReal *dp_d, hipfftReal *Dc_d, hipfftReal *Df_d, hipfftReal *Db_d, hipfftReal *Dp_d, hipfftReal *ddc, hipfftReal *ddf, hipfftReal *ddb, hipfftReal *ddp, int dM, int dD, int Nx, int Ny, int Nk, int Nl, float del, int maxdiff)
{

   // cuFFT 2D plans for kernel FFT
   hipfftHandle i_plan, f_plan;
   int rank = 2;
   int n[2] = {Nx, Ny};
   int idist = Nx*Ny, odist = Nx*(Ny/2+1);
   int inembed[] = {Nx, Ny};
   int onembed[] = {Nx, Ny/2+1};
   int istride = 1, ostride = 1;
   hipfftPlanMany(&i_plan,rank,n,onembed,ostride,odist,inembed,istride,idist,HIPFFT_C2R,dM*dD);
   hipfftPlanMany(&f_plan,rank,n,inembed,istride,idist,onembed,ostride,odist,HIPFFT_R2C,dM*dD);

   //output vector
   hipfftReal *dc_d, *df_d, *dck_d, *dfk_d;
   hipMalloc(&dc_d, dM*dD*Nx*Ny*sizeof(hipfftReal));
   hipMalloc(&df_d, dD*dM*Nx*Ny*sizeof(hipfftReal));
   hipMalloc(&dck_d, dM*dD*Nk*Nl*sizeof(hipfftReal));
   hipMalloc(&dfk_d, dD*dM*Nk*Nl*sizeof(hipfftReal));

   //Compute Inverse FFT of gradients
   hipfftExecC2R(i_plan, fdc_d, dc_d);
   hipfftExecC2R(i_plan, fdf_d, df_d);
   
   //shrink kernel
   int threads=256;
   int blocks=(dM*dD*Nk*Nl)/threads+1;
   shrink_k<<<blocks,threads>>>(dc_d, dck_d, dM, dD, Nx, Ny, Nk, Nl);
   shrink_k<<<blocks,threads>>>(df_d, dfk_d, dD, dM, Nx, Ny, Nk, Nl);

   //multiobjective optimization (min reconstruction error max filter difference)
   if(maxdiff)
   {
      hipfftReal *cd_d, *fd_d, *bd_d, *pd_d;
      hipMalloc(&cd_d, dM*dD*Nk*Nl*sizeof(hipfftReal));
      hipMalloc(&fd_d, dD*dM*Nk*Nl*sizeof(hipfftReal));
      hipMalloc(&bd_d, dM*sizeof(hipfftReal));
      hipMalloc(&pd_d, dD*sizeof(hipfftReal));
      //gradient to maximize filter difference
      gradient_diff<<<blocks,threads>>>(cd_d, fd_d, bd_d, pd_d, 
                                                  c_d, f_d, b_d, p_d, 
                                                  dD, dM, Nk, Nl);

//      hipfftReal *cd_h;//, *fd_h, *bd_h, *pd_h;
//      hipHostMalloc(&cd_h, dM*dD*Nk*Nl*sizeof(hipfftReal));
//      //hipHostMalloc(&fd_h, dD*dM*Nk*Nl*sizeof(hipfftReal));
//      //hipHostMalloc(&bd_h, dM*sizeof(hipfftReal));
//      //hipHostMalloc(&pd_h, dD*sizeof(hipfftReal));
//      hipMemcpy(cd_h, cd_d, dM*dD*Nk*Nl*sizeof(hipfftReal), hipMemcpyDeviceToHost);
//      for(int i=0;i<dM*dD*Nk*Nl;i++) cout<<cd_h[i]<<", ";
//      cout<<endl;
//      hipHostFree(cd_h);

      //update kernel values in coordinate space
      float w0=1, w1=10;
      backprop_double<<<blocks,threads>>>(c_d, f_d, b_d, p_d,  
                                     dck_d, dfk_d, db_d, dp_d,
                                     Dc_d, Df_d, Db_d, Dp_d,
                                     ddc, ddf, ddb, ddp, 
                                     cd_d, fd_d, bd_d, pd_d,
                                     dD, dM, Nk, Nl, del, w0, w1);
      hipFree(cd_d);
      hipFree(fd_d);
      hipFree(bd_d);
      hipFree(pd_d);
   }
   else
   {
      //update kernel values in coordinate space
      backprop_d<<<blocks,threads>>>(c_d, f_d, b_d, p_d,  
                                     dck_d, dfk_d, db_d, dp_d,
                                     Dc_d, Df_d, Db_d, Dp_d,
                                     ddc, ddf, ddb, ddp, 
                                     dD, dM, Nk, Nl, del);
   }
   //pad kernel
   hipMemset(dc_d, 0, dM*dD*Nx*Ny*sizeof(hipfftReal));
   hipMemset(df_d, 0, dM*dD*Nx*Ny*sizeof(hipfftReal));
   pad_k<<<blocks,threads>>>(c_d, dc_d, dM, dD, Nx, Ny, Nk, Nl);
   pad_k<<<blocks,threads>>>(f_d, df_d, dD, dM, Nx, Ny, Nk, Nl);


   //Compute FFT of updated kernel
   hipfftExecR2C(f_plan, dc_d, cfreq_d);
   hipfftExecR2C(f_plan, df_d, ffreq_d);

   hipfftDestroy(i_plan);
   hipfftDestroy(f_plan);
   hipFree(dc_d);
   hipFree(df_d);
   hipFree(dck_d);
   hipFree(dfk_d);

}

/////////////////////////////////////////////////////////////

void flatten_kernel(vector<vector<vector<vector<float> > > >& c, hipfftReal *c_d)
{

   int dM=c.size();
   int dD=c[0].size();
   int Nk=c[0][0].size();
   int Nl=c[0][0][0].size();
   int Ntot=dM*dD*Nk*Nl;

   hipfftReal *c_h;
   hipHostMalloc((void **) &c_h, Ntot*sizeof(hipfftReal));
   for(int m=0;m<dM;m++)
   {
      for(int d=0;d<dD;d++)
      {
         for(int i=0;i<Nk;i++)
         {
            for(int j=0;j<Nl;j++)
            {
              c_h[m*dD*Nk*Nl+d*Nk*Nl+i*Nl+j]=(hipfftReal)c[m][d][i][j];
            }
         }
      }
   }
   hipMemcpy(c_d, c_h, Ntot*sizeof(hipfftReal), hipMemcpyHostToDevice);
   hipHostFree(c_h);
}



/////////////////////////////////////////////////////////////
//          HOST EXTERNAL FUNCTIONS
/////////////////////////////////////////////////////////////


//run autoencoder in fft space
void autoenc_fft(vector<vector<vector<vector<float> > > >& layers, vector<vector<vector<vector<vector<float> > > > >& net_c, vector<vector<float> >& net_cfreq, vector<vector<float> >& net_b, vector<int>& scale, int fft_l)
{
   int dD=layers[0].size();
   int dM=net_c[0].size();
   int Nx=layers[0][0].size();
   int Ny=layers[0][0][0].size();
   hipfftComplex *freq_d, *ofreq_d, *cfreq_d;
   hipfftReal *b_d;
   hipMalloc(&freq_d, dD*Nx*(Ny/2+1)*sizeof(hipfftComplex));
   fft(layers[0],freq_d);
   int l=1;
   for(int n=0;n<net_c.size();n++)
   {
      if(n<net_c.size()/2) 
      {
         pool_fft(freq_d, dD, Nx, Ny, scale[n]);
         if(fft_l) {fft_inv(freq_d,layers[l]); l+=1;}
      }
      hipMalloc(&ofreq_d, dM*Nx*(Ny/2+1)*sizeof(hipfftComplex));
      hipMalloc(&cfreq_d, dM*dD*Nx*(Ny/2+1)*sizeof(hipfftComplex));
      hipMalloc(&b_d, dM*sizeof(hipfftReal));
      hipMemset(cfreq_d, 0, dM*dD*Nx*(Ny/2+1)*sizeof(hipfftComplex));
      StoreLoad_cfreq(net_cfreq, net_c, net_b, cfreq_d, b_d, dM, dD, Nx, Ny, n);
//hipMemcpy(ofreq_d, freq_d, dM*Nx*(Ny/2+1)*sizeof(hipfftComplex), 
//                  hipMemcpyDeviceToDevice);
      conv_fft(freq_d, ofreq_d, cfreq_d, b_d, dM, dD, Nx, Ny);
      if(fft_l) {fft_inv(ofreq_d,layers[l]); l+=1;}
      if(n>=net_c.size()/2) 
      {
         pool_fft(ofreq_d, dM, Nx, Ny, scale[n]);
         if(fft_l) {fft_inv(ofreq_d,layers[l]); l+=1;}
      }
      hipFree(freq_d);
      hipMalloc(&freq_d, dM*Nx*(Ny/2+1)*sizeof(hipfftComplex));
      hipMemcpy(freq_d, ofreq_d, dM*Nx*(Ny/2+1)*sizeof(hipfftComplex), 
                  hipMemcpyDeviceToDevice);
      dD=dM;
      dM=net_c[n+1].size();
      hipFree(ofreq_d);
      hipFree(cfreq_d);
      hipFree(b_d);
   }
   if(!fft_l) fft_inv(freq_d,layers.back());
   hipFree(freq_d);

}

/////////////////////////////////////////////////////////////

//run backpropagation in fft space
void backprop_fft(vector<vector<vector<float> > >& in, vector<vector<vector<float> > >& out, vector<float>& cfreq, vector<vector<vector<vector<float> > > >& c, vector<float>& ffreq, vector<vector<vector<vector<float> > > >& f, vector<float>& b, vector<float>& p, int dM, float del0, int maxdiff)
{
   int dD=in.size();
   int Nx=in[0].size();
   int Ny=in[0][0].size();
   int Nk=c[0][0].size();
   int Nl=c[0][0][0].size();
   hipfftComplex *freq_d, *hfreq_d, *ofreq_d, *cfreq_d, *ffreq_d;
   hipfftReal *c_d, *f_d, *b_d, *p_d;
   hipfftReal *Dc_d, *Df_d, *Db_d, *Dp_d;
   hipfftReal *ddc, *ddf, *ddb, *ddp;
   hipfftComplex *dc_d, *df_d;
   hipfftReal *db_d, *dp_d;
   hipMalloc(&freq_d, dD*Nx*(Ny/2+1)*sizeof(hipfftComplex));
   hipMalloc(&hfreq_d, dM*Nx*(Ny/2+1)*sizeof(hipfftComplex));
   hipMalloc(&ofreq_d, dD*Nx*(Ny/2+1)*sizeof(hipfftComplex));
   hipMalloc(&cfreq_d, dM*dD*Nx*(Ny/2+1)*sizeof(hipfftComplex));
   hipMalloc(&ffreq_d, dD*dM*Nx*(Ny/2+1)*sizeof(hipfftComplex));
   hipMalloc(&c_d, dM*dD*Nk*Nl*sizeof(hipfftReal));
   hipMalloc(&f_d, dD*dM*Nk*Nl*sizeof(hipfftReal));
   hipMalloc(&b_d, dM*sizeof(hipfftReal));
   hipMalloc(&p_d, dD*sizeof(hipfftReal));
   hipMalloc(&dc_d, dM*dD*Nx*(Ny/2+1)*sizeof(hipfftComplex));
   hipMalloc(&df_d, dD*dM*Nx*(Ny/2+1)*sizeof(hipfftComplex));
   hipMalloc(&db_d, dM*sizeof(hipfftReal));
   hipMalloc(&dp_d, dD*sizeof(hipfftReal));
   hipMalloc(&Dc_d, dM*dD*Nk*Nl*sizeof(hipfftReal));
   hipMalloc(&Df_d, dD*dM*Nk*Nl*sizeof(hipfftReal));
   hipMalloc(&Db_d, dM*sizeof(hipfftReal));
   hipMalloc(&Dp_d, dD*sizeof(hipfftReal));
   hipMalloc(&ddc, dM*dD*Nk*Nl*sizeof(hipfftReal));
   hipMalloc(&ddf, dD*dM*Nk*Nl*sizeof(hipfftReal));
   hipMalloc(&ddb, dM*sizeof(hipfftReal));
   hipMalloc(&ddp, dD*sizeof(hipfftReal));
   hipMemset(dc_d, 0, dM*dD*Nx*(Ny/2+1)*sizeof(hipfftComplex));
   hipMemset(df_d, 0, dD*dM*Nx*(Ny/2+1)*sizeof(hipfftComplex));
   hipMemset(db_d, 0, dM*sizeof(hipfftReal));
   hipMemset(dp_d, 0, dD*sizeof(hipfftReal));
   hipMemset(Dc_d, 0, dM*dD*Nk*Nl*sizeof(hipfftReal));
   hipMemset(Df_d, 0, dD*dM*Nk*Nl*sizeof(hipfftReal));
   hipMemset(Db_d, 0, dM*sizeof(hipfftReal));
   hipMemset(Dp_d, 0, dD*sizeof(hipfftReal));
   hipMemset(ddc, 0, dM*dD*Nk*Nl*sizeof(hipfftReal));
   hipMemset(ddf, 0, dD*dM*Nk*Nl*sizeof(hipfftReal));
   hipMemset(ddb, 0, dM*sizeof(hipfftReal));
   hipMemset(ddp, 0, dD*sizeof(hipfftReal));

   //fft in out
   fft(in,freq_d);
   fft(out,ofreq_d);
   //load fft conv weights
   load_cfreq(cfreq, b, cfreq_d, b_d, dM, dD, Nx, Ny/2+1);
   load_cfreq(ffreq, p, ffreq_d, p_d, dD, dM, Nx, Ny/2+1);
   flatten_kernel(c,c_d);
   flatten_kernel(f,f_d);

   //mse fft
   float vmse=mse_fft(freq_d, ofreq_d, dM, dD, Nx, Ny);
   cout<<"mse fft: "<<vmse<<endl;
   //float vmse_prev=vmse;
   //backpropagation
   //float del=0.00002;
   float del=0.01*del0;
   for(int n=0;n<100;n++)
   {
      int threads=256;
      int blocks=(dM*dD*Nx*(Ny/2+1))/threads+1;
//      backprop_k<<<blocks,threads>>>(freq_d, ofreq_d, cfreq_d, ffreq_d, b_d, p_d, 
//                                    cfreq1_d, ffreq1_d, b1_d, p1_d, 
//                                    dc_d, df_d, db_d, dp_d, 
//                                    ddc_d, ddf_d, ddb_d, ddp_d, 
//                                    dM, dD, Nx, Ny);
      //float del=del0;
      //if(vmse<10) del=10*del0;
      gradient_k<<<blocks,threads>>>(freq_d, ofreq_d, cfreq_d, ffreq_d, b_d, p_d,  
                                    dc_d, df_d, db_d, dp_d, dM, dD, Nx, Ny);
      backprop(c_d, f_d, cfreq_d, ffreq_d, b_d, p_d, dc_d, df_d, db_d, dp_d, 
               Dc_d, Df_d, Db_d, Dp_d, ddc, ddf, ddb, ddp, 
               dM, dD, Nx, Ny, Nk, Nl, del, maxdiff);

      conv_fft(freq_d, hfreq_d, cfreq_d, b_d, dM, dD, Nx, Ny);
      conv_fft(hfreq_d, ofreq_d, ffreq_d, p_d, dD, dM, Nx, Ny);
      float vmse=mse_fft(freq_d, ofreq_d, dM, dD, Nx, Ny);
      cout<<"n: "<<n<<" mse: "<<vmse<<endl;
//      if(vmse<vmse_prev)
//      {
//         hipMemcpy(cfreq_d, cfreq1_d, dM*dD*Nx*(Ny/2+1)*sizeof(hipfftComplex), 
//                     hipMemcpyDeviceToDevice);
//         hipMemcpy(ffreq_d, ffreq1_d, dM*dD*Nx*(Ny/2+1)*sizeof(hipfftComplex), 
//                     hipMemcpyDeviceToDevice);
//         hipMemcpy(b_d, b1_d, dM*sizeof(hipfftReal), 
//                     hipMemcpyDeviceToDevice);
//         hipMemcpy(p_d, p1_d, dD*sizeof(hipfftReal), 
//                     hipMemcpyDeviceToDevice);
//         vmse_prev=vmse;
//      }
//      ofstream file;
//      file.open("./fmse.txt", ios::out|ios::app);
//      file<<setprecision(9)<<vmse<<"\n"; 
//      file.close();
   }
   //inverse fft and store learned weights
   //fft_inv(ofreq_d,out);
   store_cfreq(cfreq_d, cfreq, dM, dD, Nx, Ny/2+1);
   store_cfreq(ffreq_d, ffreq, dD, dM, Nx, Ny/2+1);
   //export learned kernels
   export_cfreq(c, b, cfreq_d, b_d, dM, dD, Nx, Ny);
   export_cfreq(f, p, ffreq_d, p_d, dD, dM, Nx, Ny);
   hipFree(freq_d);
   hipFree(hfreq_d);
   hipFree(ofreq_d);
   hipFree(cfreq_d);
   hipFree(ffreq_d);
   hipFree(c_d);
   hipFree(f_d);
   hipFree(b_d);
   hipFree(p_d);
   hipFree(dc_d);
   hipFree(df_d);
   hipFree(db_d);
   hipFree(dp_d);
   hipFree(Dc_d);
   hipFree(Df_d);
   hipFree(Db_d);
   hipFree(Dp_d);
   hipFree(ddc);
   hipFree(ddf);
   hipFree(ddb);
   hipFree(ddp);
}

